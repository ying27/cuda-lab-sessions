#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define SIZE 32

#ifndef PINNED
#define PINNED 0
#endif


// Matriz por Matriz
// C(NxM) <- A(NxP) * B (PxM)

__global__ void Kernel01 (int N, int M, int P, float *A, float *B, float *C) {

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < N && col < M) {
    float tmp = 0.0;
    for (int k=0; k<P; k++)
      tmp += A[row*P+k] * B[k*M+col];
    C[row*M+col] = tmp;
  }
}



void InitM(int N, int M, float *Mat);
int TestMM(int N, int M, int P, float *A, float *B, float *C);


// Invocacion:
// ./ejecutable TAM test
// TAM es el la dimension de las matrices
// test == 'Y', comprueba que el resultado sea correcto
// test == 'N', NO comprueba que el resultado (Util para tomar tiempos)
// Por defecto, N = 2048, test == 'N'

int main(int argc, char** argv)
{
  unsigned int N;
  unsigned int numBytes;
  unsigned int nBlocks, nThreads;
 
  float TiempoTotal, TiempoKernel;
  hipEvent_t E0, E1, E2, E3;

  float *h_A, *h_B, *h_C;
  float *d_A, *d_B, *d_C;

  char test;

  // Dimension de las matrices NxN y comprobacion resultado
  if (argc == 1)      { test = 'N'; N = 2048; }
  else if (argc == 2) { test = 'N'; N = atoi(argv[1]); }
  else if (argc == 3) { test = *argv[2]; N = atoi(argv[1]); }
  else { printf("Usage: ./exe TAM test\n"); exit(0); }

  // numero de Threads en cada dimension 
  nThreads = SIZE;

  // numero de Blocks en cada dimension 
  nBlocks = (N+nThreads-1)/nThreads; 
  
  numBytes = N * N * sizeof(float);

  dim3 dimGrid(nBlocks, nBlocks, 1);
  dim3 dimBlock(nThreads, nThreads, 1);

  hipEventCreate(&E0);
  hipEventCreate(&E1);
  hipEventCreate(&E2);
  hipEventCreate(&E3);

  if (PINNED) {
    // Obtiene Memoria [pinned] en el host
    hipHostMalloc((float**)&h_A, numBytes); 
    hipHostMalloc((float**)&h_B, numBytes); 
    hipHostMalloc((float**)&h_C, numBytes); 
  }
  else {
    // Obtener Memoria en el host
    h_A = (float*) malloc(numBytes); 
    h_B = (float*) malloc(numBytes); 
    h_C = (float*) malloc(numBytes); 
  }

  // Inicializa las matrices
  InitM(N, N, h_A);
  InitM(N, N, h_B);

  hipEventRecord(E0, 0);
  hipEventSynchronize(E0);
  
  // Obtener Memoria en el device
  hipMalloc((float**)&d_A, numBytes); 
  hipMalloc((float**)&d_B, numBytes); 
  hipMalloc((float**)&d_C, numBytes); 

  // Copiar datos desde el host en el device 
  hipMemcpy(d_A, h_A, numBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, numBytes, hipMemcpyHostToDevice);

  hipEventRecord(E1, 0);
  hipEventSynchronize(E1);
  
  // Ejecutar el kernel 
  Kernel01<<<dimGrid, dimBlock>>>(N, N, N, d_A, d_B, d_C);

  hipEventRecord(E2, 0);
  hipEventSynchronize(E2);

  // Obtener el resultado desde el host 
  hipMemcpy(h_C, d_C, numBytes, hipMemcpyDeviceToHost); 

  // Liberar Memoria del device 
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  hipEventRecord(E3, 0);
  hipEventSynchronize(E3);

  hipEventElapsedTime(&TiempoTotal,  E0, E3);
  hipEventElapsedTime(&TiempoKernel, E1, E2);
  printf("\nKERNEL 01\n");
  printf("Dimensiones: %dx%d\n", N, N);
  printf("nThreads: %dx%d (%d)\n", nThreads, nThreads, nThreads * nThreads);
  printf("nBlocks: %dx%d (%d)\n", nBlocks, nBlocks, nBlocks*nBlocks);
  if (PINNED) printf("Usando Pinned Memory\n");
         else printf("NO usa Pinned Memory\n");
  printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
  printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);
  printf("Rendimiento Global: %4.2f GFLOPS\n", (2.0 * (float) N * (float) N * (float) N) / (1000000.0 * TiempoTotal));
  printf("Rendimiento Kernel: %4.2f GFLOPS\n", (2.0 * (float) N * (float) N * (float) N) / (1000000.0 * TiempoKernel));

  hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);

  if (test == 'N')
    printf ("NO TEST\n");
  else  if (TestMM(N, N, N, h_A, h_B, h_C))
    printf ("TEST PASS\n");
  else
    printf ("TEST FAIL\n");

  if (PINNED) {
    hipHostFree(h_A); hipHostFree(h_B); hipHostFree(h_C);
  }
  else {
    free(h_A); free(h_B); free(h_C);
  }

}


void InitM(int N, int M, float *Mat) {
   int i;
   for (i=0; i<N*M; i++) 
     Mat[i] = rand() / (float) RAND_MAX;
   
}

int error(float a, float b) {
  float tmp;

  tmp = abs(a-b) / abs(min(a,b));

  if (tmp > 0.0001) return 1;
  else  return 0;

}

int TestMM(int N, int M, int P, float *A, float *B, float *C) {
   int i, j, k;
   float tmp;
   for (i=0; i<N; i++)
     for (j=0; j<M; j++) {
       tmp = 0.0;
       for (k=0; k<P; k++) 
         tmp = tmp + A[i*P+k] * B[k*M+j]; 
       if (error(tmp, C[i*M+j])) {
         printf ("%d:%d: %f - %f = %f \n", i, j, tmp, C[i*M+j], abs(tmp - C[i*M+j]));
         return 0;
       }
     }
   
   return 1;
}

