#include "hip/hip_runtime.h"
/** @file histo-global.cu histogram with global memory atomics */

#include <png.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

/** CUDA check macro */
#define cucheck(call) \
	{\
	hipError_t res = (call);\
	if(res != hipSuccess) {\
	const char* err_str = hipGetErrorString(res);\
	fprintf(stderr, "%s (%d): %s in %s", __FILE__, __LINE__, err_str, #call);	\
	exit(-1);\
	}\
	}

/** time spent in device */
double gpu_time = 0;

/** a useful function to compute the number of threads */
int divup(int x, int y) { return x / y + (x % y ? 1 : 0); }

/** gets the color, given the dwell */
void dwell_color(int *r, int *g, int *b, int dwell);

/** save the dwell into a PNG file 
		@remarks: code to save PNG file taken from here 
		  (error handling is removed):
		http://www.labbookpages.co.uk/software/imgProc/libPNG.html
 */
void save_image(const char *filename, int *dwells, int w, int h) {
	png_bytep row;
	
	FILE *fp = fopen(filename, "wb");
	png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, 0, 0, 0);
	png_infop info_ptr = png_create_info_struct(png_ptr);
	// exception handling
	setjmp(png_jmpbuf(png_ptr));
	png_init_io(png_ptr, fp);
	// write header (8 bit colour depth)
	png_set_IHDR(png_ptr, info_ptr, w, h,
							 8, PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
							 PNG_COMPRESSION_TYPE_BASE, PNG_FILTER_TYPE_BASE);
	// set title
	png_text title_text;
	title_text.compression = PNG_TEXT_COMPRESSION_NONE;
	title_text.key = "Title";
	title_text.text = "Mandelbrot set, per-pixel";
	png_set_text(png_ptr, info_ptr, &title_text, 1);
	png_write_info(png_ptr, info_ptr);

	// write image data
	row = (png_bytep) malloc(3 * w * sizeof(png_byte));
	for (int y = 0; y < h; y++) {
		for (int x = 0; x < w; x++) {
			int r, g, b;
			dwell_color(&r, &g, &b, dwells[y * w + x]);
			row[3 * x + 0] = (png_byte)r;
			row[3 * x + 1] = (png_byte)g;
			row[3 * x + 2] = (png_byte)b;
		}
		png_write_row(png_ptr, row);
	}
	png_write_end(png_ptr, NULL);

  fclose(fp);
  png_free_data(png_ptr, info_ptr, PNG_FREE_ALL, -1);
  png_destroy_write_struct(&png_ptr, (png_infopp)NULL);
  free(row);
}  // save_image


/** a simple complex type */
struct complex {
	__host__ __device__ complex(float re, float im = 0) {
		this->re = re;
		this->im = im;
	}
	/** real and imaginary part */
	float re, im;
}; // struct complex

// operator overloads for complex numbers
inline __host__ __device__ complex operator+
(const complex &a, const complex &b) {
	return complex(a.re + b.re, a.im + b.im);
}
inline __host__ __device__ complex operator-
(const complex &a) { return complex(-a.re, -a.im); }
inline __host__ __device__ complex operator-
(const complex &a, const complex &b) {
	return complex(a.re - b.re, a.im - b.im);
}
inline __host__ __device__ complex operator*
(const complex &a, const complex &b) {
	return complex(a.re * b.re - a.im * b.im, a.im * b.re + a.re * b.im);
}
inline __host__ __device__ float abs2(const complex &a) {
	return a.re * a.re + a.im * a.im;
}
inline __host__ __device__ complex operator/
(const complex &a, const complex &b) {
	float invabs2 = 1 / abs2(b);
	return complex((a.re * b.re + a.im * b.im) * invabs2,
								 (a.im * b.re - b.im * a.re) * invabs2);
}  // operator/

#define MAX_DWELL 256
#define BS 256
/** computes the dwell for a single pixel */
__device__ int pixel_dwell
(int w, int h, complex cmin, complex cmax, int x, int y) {
	complex dc = cmax - cmin;
	float fx = (float)x / w, fy = (float)y / h;
	complex c = cmin + complex(fx * dc.re, fy * dc.im);
	int dwell = 0;
	complex z = c;
	while(dwell < MAX_DWELL && abs2(z) < 2 * 2) {
		z = z * z + c;
		dwell++;
	}
	return dwell;
}  // pixel_dwell

/** computes the dwells for Mandelbrot image 
		@param dwells the output array
		@param w the width of the output image
		@param h the height of the output image
		@param cmin the complex value associated with the left-bottom corner of the
		image
		@param cmax the complex value associated with the right-top corner of the
		image
 */
__global__ void mandelbrot_k
(int *dwells, int w, int h, complex cmin, complex cmax) {
	// complex value to start iteration (c)
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int dwell = pixel_dwell(w, h, cmin, cmax, x, y);
	dwells[y * w + x] = dwell;
}  // mandelbrot_k

/** gets the color, given the dwell (on host) */
#define CUT_DWELL (MAX_DWELL / 4)
void dwell_color(int *r, int *g, int *b, int dwell) {
	// black for the Mandelbrot set
	if(dwell >= MAX_DWELL) {
		*r = *g = *b = 0;
	} else {
		// cut at zero
		if(dwell < 0)
			dwell = 0;
		if(dwell <= CUT_DWELL) {
			// from black to blue the first half
			*r = *g = 0;
			*b = 128 + dwell * 127 / (CUT_DWELL);
		} else {
			// from blue to white for the second half
			*b = 255;
			*r = *g = (dwell - CUT_DWELL) * 255 / (MAX_DWELL - CUT_DWELL);
		}
	}
}  // dwell_color

/** data size */
#define H (8 * 1024)
#define W (8 * 1024)
#define IMAGE_PATH "./mandelbrot.png"

int main(int argc, char **argv) {
	// allocate memory
	int w = W, h = H;
	size_t dwell_sz = w * h * sizeof(int);
	int *h_dwells, *d_dwells;
	cucheck(hipMalloc((void**)&d_dwells, dwell_sz));
	h_dwells = (int*)malloc(dwell_sz);

	// compute the dwells, copy them back
	double t1 = omp_get_wtime();
	dim3 bs(64, 4), grid(divup(w, bs.x), divup(h, bs.y));
	mandelbrot_k<<<grid, bs>>>
		(d_dwells, w, h, complex(-1.5, -1), complex(0.5, 1));
	cucheck(hipDeviceSynchronize());
	double t2 = omp_get_wtime();
	cucheck(hipMemcpy(h_dwells, d_dwells, dwell_sz, hipMemcpyDeviceToHost));
	gpu_time = t2 - t1;
	
	// save the image to PNG 
	save_image(IMAGE_PATH, h_dwells, w, h);

	// print performance
	printf("Mandelbrot set computed in %.3lf s, at %.3lf Mpix/s\n", gpu_time, 
				 h * w * 1e-6 / gpu_time);

	// free data
	hipFree(d_dwells);
	free(h_dwells);
	return 0;
}  // main
