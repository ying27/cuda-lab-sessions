#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#ifndef SIZE
#define SIZE 32
#endif

// Kernel Matriz por Matriz
// C(NxM) <- A(NxP) * B (PxM)

__global__ void KernelMM(int N, int M, int P, float *A, float *B, float *C) {

  __shared__ float sA[SIZE][SIZE];
  __shared__ float sB[SIZE][SIZE];

  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = by * SIZE + ty;
  int col = bx * SIZE + tx;

  float tmp = 0.0;
  for (int m=0; m < P; m=m+SIZE) {
    sA[ty][tx] = A[row*P + m + tx];
    sB[ty][tx] = B[col + (m + ty)*M];
    __syncthreads();
    for (int k=0; k<SIZE; k++)
      tmp += sA[ty][k] * sB[k][tx];
    __syncthreads();
  }
  C[row*M+col] = tmp;
}



void InitM(int N, int M, float *Mat);
int TestMM(int N, int M, int P, float *A, float *B, float *C);

int nTest = 0;

// Invocacion:
// ./ejecutable TAM test
// TAM es el la dimension de las matrices
// test == 'Y', comprueba que el resultado sea correcto
// test == 'N', NO comprueba que el resultado (Util para tomar tiempos)
// Por defecto, N = 2048, test == 'N'

int main(int argc, char** argv)
{
  unsigned int N;
  unsigned int numBytesC, numBytesA, numBytesB;
  unsigned int nBlocks, nThreads;
 
  float TiempoTotal, TiempoKernel;
  hipEvent_t E0, E1, E2, E3;
  hipEvent_t X1, X2, X3;

  float *hA0, *hA1, *hB0, *hB1, *hC00, *hC01, *hC10, *hC11;
  float *dA0a, *dA1a, *dB0a, *dB1a, *dC00, *dC01, *dC10, *dC11;
  float *dA0b, *dA1b, *dB0b, *dB1b;

  int count;

  char test;

  // Dimension de las matrices NxN y comprobacion resultado
  if (argc == 1)      { test = 'N'; N = 2048; }
  else if (argc == 2) { test = 'N'; N = atoi(argv[1]); }
  else if (argc == 3) { test = *argv[2]; N = atoi(argv[1]); }
  else { printf("Usage: ./exe TAM test\n"); exit(0); }

  // numero de Threads en cada dimension 
  nThreads = SIZE;

  // numero de Blocks en cada dimension 
  nBlocks = (N/2)/nThreads; 
  
  numBytesC = N * N * sizeof(float) / 4;
  numBytesA = N * N * sizeof(float) / 2;
  numBytesB = N * N * sizeof(float) / 2;

  dim3 dimGrid(nBlocks, nBlocks, 1);
  dim3 dimBlock(nThreads, nThreads, 1);


  hipGetDeviceCount(&count);

  if (count < 4) { printf("No hay suficientes GPUs\n"); exit(0); }

  // Obtiene Memoria [pinned] en el host
  hipHostMalloc((float**)&hA0,  numBytesA); 
  hipHostMalloc((float**)&hA1,  numBytesA); 
  hipHostMalloc((float**)&hB0,  numBytesB); 
  hipHostMalloc((float**)&hB1,  numBytesB); 
  hipHostMalloc((float**)&hC00, numBytesC); 
  hipHostMalloc((float**)&hC01, numBytesC); 
  hipHostMalloc((float**)&hC10, numBytesC); 
  hipHostMalloc((float**)&hC11, numBytesC); 

  // Inicializa las matrices
  InitM(N/2, N, hA0);
  InitM(N/2, N, hA1);
  InitM(N, N/2, hB0);
  InitM(N, N/2, hB1);


  // Obtener Memoria en cada device
  hipSetDevice(0);
  hipMalloc((float**)&dA0a, numBytesA); 
  hipMalloc((float**)&dB0a, numBytesB); 
  hipMalloc((float**)&dC00, numBytesC); 

  hipSetDevice(1);
  hipMalloc((float**)&dA0b, numBytesA); 
  hipMalloc((float**)&dB1a, numBytesB); 
  hipMalloc((float**)&dC01, numBytesC); 
  hipEventCreate(&X1); 

  hipSetDevice(2);
  hipMalloc((float**)&dA1a, numBytesA); 
  hipMalloc((float**)&dB0b, numBytesB); 
  hipMalloc((float**)&dC10, numBytesC); 
  hipEventCreate(&X2);

  hipSetDevice(3);
  hipMalloc((float**)&dA1b, numBytesA); 
  hipMalloc((float**)&dB1b, numBytesB); 
  hipMalloc((float**)&dC11, numBytesC); 
  hipEventCreate(&X3);

  hipSetDevice(0);
  hipEventCreate(&E0);
  hipEventCreate(&E1);
  hipEventCreate(&E2);
  hipEventCreate(&E3);

  hipEventRecord(E0, 0);

  // Copiar datos desde el host en el device 
  hipMemcpyAsync(dA0a, hA0, numBytesA, hipMemcpyHostToDevice);
  hipMemcpyAsync(dB0a, hB0, numBytesB, hipMemcpyHostToDevice);
  hipEventRecord(E1, 0); 
  // Ejecutar el kernel 
  KernelMM<<<dimGrid, dimBlock>>>(N/2, N/2, N, dA0a, dB0a, dC00);
  hipEventRecord(E2, 0); hipEventSynchronize(E2);
  // Obtener el resultado desde el host 
  hipMemcpyAsync(hC00, dC00, numBytesC, hipMemcpyDeviceToHost); 

  hipSetDevice(1);
  // Copiar datos desde el host en el device 
  hipMemcpyAsync(dA0b, hA0, numBytesA, hipMemcpyHostToDevice);
  hipMemcpyAsync(dB1a, hB1, numBytesB, hipMemcpyHostToDevice);
  // Ejecutar el kernel 
  KernelMM<<<dimGrid, dimBlock>>>(N/2, N/2, N, dA0b, dB1a, dC01);
  // Obtener el resultado desde el host 
  hipMemcpyAsync(hC01, dC01, numBytesC, hipMemcpyDeviceToHost); 
  hipEventRecord(X1, 0);

  hipSetDevice(2);
  // Copiar datos desde el host en el device 
  hipMemcpyAsync(dA1a, hA1, numBytesA, hipMemcpyHostToDevice);
  hipMemcpyAsync(dB0b, hB0, numBytesB, hipMemcpyHostToDevice);
  // Ejecutar el kernel 
  KernelMM<<<dimGrid, dimBlock>>>(N/2, N/2, N, dA1a, dB0b, dC10);
  // Obtener el resultado desde el host 
  hipMemcpyAsync(hC10, dC10, numBytesC, hipMemcpyDeviceToHost); 
  hipEventRecord(X2, 0);

  hipSetDevice(3);
  // Copiar datos desde el host en el device 
  hipMemcpyAsync(dA1b, hA1, numBytesA, hipMemcpyHostToDevice);
  hipMemcpyAsync(dB1b, hB1, numBytesB, hipMemcpyHostToDevice);
  // Ejecutar el kernel 
  KernelMM<<<dimGrid, dimBlock>>>(N/2, N/2, N, dA1b, dB1b, dC11);
  // Obtener el resultado desde el host 
  hipMemcpyAsync(hC11, dC11, numBytesC, hipMemcpyDeviceToHost); 
  hipEventRecord(X3, 0);


  hipSetDevice(0);
  hipEventSynchronize(X1);
  hipEventSynchronize(X2);
  hipEventSynchronize(X3);

  hipEventRecord(E3, 0); hipEventSynchronize(E3);

  // Liberar Memoria del device 
  hipSetDevice(0); hipFree(dA0a); hipFree(dB0a); hipFree(dC00); 
  hipSetDevice(1); hipFree(dA0b); hipFree(dB1a); hipFree(dC01); 
  hipSetDevice(2); hipFree(dA1a); hipFree(dB0b); hipFree(dC10); 
  hipSetDevice(3); hipFree(dA1b); hipFree(dB1b); hipFree(dC11); 

  hipEventElapsedTime(&TiempoTotal,  E0, E3);
  hipEventElapsedTime(&TiempoKernel, E1, E2);
  printf("\nKERNEL MultiGPU - Producto Matrices\n");
  printf("Dimensiones: %dx%d\n", N, N);
  printf("nThreads: %dx%d (%d)\n", nThreads, nThreads, nThreads * nThreads);
  printf("nBlocks: %dx%d (%d)\n", nBlocks, nBlocks, nBlocks*nBlocks);
  printf("Usando Pinned Memory\n");
  printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
  printf("Tiempo 1 Kernel: %4.6f milseg\n", TiempoKernel);
  printf("Rendimiento Global: %4.2f GFLOPS\n", (2.0 * (float) N * (float) N * (float) N) / (1000000.0 * TiempoTotal));
  printf("Rendimiento 1 Kernel:  %4.2f GFLOPS\n", (0.5 * (float) N * (float) N * (float) N) / (1000000.0 * TiempoKernel));
  printf("Rendimiento 4 Kernels: %4.2f GFLOPS\n", (2.0 * (float) N * (float) N * (float) N) / (1000000.0 * TiempoKernel));

  hipSetDevice(0); hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);
  hipSetDevice(1); hipEventDestroy(X1);
  hipSetDevice(2); hipEventDestroy(X2);
  hipSetDevice(3); hipEventDestroy(X3);

  if (test == 'N')
    printf ("NO TEST\n");
  else  if (TestMM(N/2, N/2, N, hA0, hB0, hC00) && 
            TestMM(N/2, N/2, N, hA0, hB1, hC01) && 
            TestMM(N/2, N/2, N, hA1, hB0, hC10) &&
            TestMM(N/2, N/2, N, hA1, hB1, hC11)) 
    printf ("TEST PASS\n");
  else
    printf ("TEST FAIL\n");

  hipHostFree(hA0); hipHostFree(hA1); 
  hipHostFree(hB0); hipHostFree(hB1); 
  hipHostFree(hC00); hipHostFree(hC01); hipHostFree(hC10); hipHostFree(hC11);

}


void InitM(int N, int M, float *Mat) {
   int i;
   for (i=0; i<N*M; i++) 
     Mat[i] = rand() / (float) RAND_MAX;
   
}

int error(float a, float b) {
  float tmp;

  tmp = abs(a-b) / abs(min(a,b));

  if (tmp > 0.0001) return 1;
  else  return 0;

}

int TestMM(int N, int M, int P, float *A, float *B, float *C) {
   int i, j, k;
   float tmp;
   printf("Pass %d\n", nTest); nTest++;
   for (i=0; i<N; i++)
     for (j=0; j<M; j++) {
       tmp = 0.0;
       for (k=0; k<P; k++) 
         tmp = tmp + A[i*P+k] * B[k*M+j]; 
       if (error(tmp, C[i*M+j])) {
         printf ("%d:%d: %f - %f = %f \n", i, j, tmp, C[i*M+j], abs(tmp - C[i*M+j]));
         return 0;
       }
     }
   
   return 1;
}

